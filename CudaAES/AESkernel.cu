#include "hip/hip_runtime.h"
﻿// main.cpp
#include "AES.cuh"

#include <chrono>
#include <cstdlib>



// Get File Name
std::string GetFileName(const std::string& filePath) {
    size_t lastSlash = filePath.find_last_of("/\\");
    if (lastSlash == std::string::npos) {
        return filePath;  // No slashes, return the whole string
    }
    return filePath.substr(lastSlash + 1);  // Return everything after the last slash
}

// Create file path (if dont exist)
std::string CreateFilePath(const std::string& folder, const std::string& fileName) {
    return folder + "/" + fileName;
}

// Function to append "_encrypted" or "_decrypted" to the file
std::string AppendToFileName(const std::string& filePath, const std::string& suffix) {
    size_t dotPosition = filePath.find_last_of('.');
    if (dotPosition == std::string::npos) {
        return filePath + suffix;  // No file extension, just append the suffix
    }
    else {
        return filePath.substr(0, dotPosition) + suffix + filePath.substr(dotPosition);  // Insert suffix before extension
    }
}

// remove the key if longer than 32 characters, or pad it with zero if shorter
void PadOrTruncateKey(unsigned char* key, const std::string& inputKey) {
    std::memset(key, 0, SIZE_32);  
    std::memcpy(key, inputKey.c_str(), std::min(inputKey.length(), static_cast<size_t>(SIZE_32)));
}

void DisplayAESExplanationkey(unsigned char* key) {
    const int expandedKeySizeDisplay = 240;

    // the expanded key
    unsigned char expandedKeyDisplay[expandedKeySizeDisplay];

    CreateExpandKey(expandedKeyDisplay, key, SIZE_32, expandedKeySizeDisplay);

    std::cout << "Expanded Key:\n";
    for (int i = 0; i < expandedKeySizeDisplay; i++) {
       
        if (i % 16 == 0) {
            std::cout << (i / 16 + 1) << ": ";  
        }

        std::cout << std::hex << std::setw(2) << std::setfill('0')
            << static_cast<int>(expandedKeyDisplay[i]);

        
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;  
        }
        else {
            std::cout << " ";  
        }
    }

}

bool FileExists(const std::string& path) {
    std::ifstream file(path);
    return file.good();
}

bool AskForKey(unsigned char* key) {
    std::string inputKey;
    std::cout << "Enter the AES key (up to 32 characters for AES-256): ";
    std::cin >> inputKey;

    if (inputKey.length() > SIZE_32) {
        std::cerr << "Key exceeds 32 characters, it will be truncated to 32 characters." << std::endl;
    }

  
    PadOrTruncateKey(key, inputKey);
    DisplayAESExplanationkey(key);
    std::cout << "\n";
    return true;
}

bool AskForFilePath(std::string& fileName) {
    std::cout << "Enter the file name in the 'FileToEncrypt' folder (e.g., example.txt): ";
    std::cin >> fileName;

    std::string inputFilePath = CreateFilePath("FileToEncrypt", fileName);
    if (!FileExists(inputFilePath)) {
        std::cerr << "File does not exist in 'FileToEncrypt' folder. Please try again." << std::endl;
        return false;
    }

    return true;
}

void EncryptProcess() {
    std::string fileName;
    unsigned char key[SIZE_32];

    // Ask for user the file name and key
    if (!AskForFilePath(fileName) || !AskForKey(key)) {
        return;
    }

    
    std::string inputFilePath = CreateFilePath("FileToEncrypt", fileName);
    std::string encryptedFilePath = CreateFilePath("EncryptFile", AppendToFileName(fileName, "_encrypted"));

    
    auto encryptionStart = std::chrono::high_resolution_clock::now();

    
    if (EncryptFile(inputFilePath, encryptedFilePath, key, SIZE_32)) {
        std::cout << "File encryption completed successfully! Encrypted file stored at: " << encryptedFilePath << std::endl;
    }
    else {
        std::cerr << "File encryption failed!" << std::endl;
        return;
    }

    auto encryptionEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> encryptionDuration = encryptionEnd - encryptionStart;
    std::cout << "Time taken for encryption: " << encryptionDuration.count() << " seconds" << std::endl;
}

void DecryptProcess() {
    std::string fileName;
    unsigned char key[SIZE_32];

    
    std::cout << "Enter the encrypted file name (e.g., example_encrypted.bin): ";
    std::cin >> fileName;

    // Check file exist or not
    std::string encryptedFilePath = CreateFilePath("EncryptFile", fileName);
    if (!FileExists(encryptedFilePath)) {
        std::cerr << "File does not exist in 'EncryptFile' folder. Please try again." << std::endl;
        return;
    }

    std::string decryptedFilePath = CreateFilePath("DecryptFile", AppendToFileName(GetFileName(fileName), "_decrypted"));

    // Ask for user key
    if (!AskForKey(key)) {
        return;
    }

   
    auto decryptionStart = std::chrono::high_resolution_clock::now();

    
    if (DecryptFile(encryptedFilePath, decryptedFilePath, key, SIZE_32)) {
        std::cout << "File decryption completed successfully! Decrypted file stored at: " << decryptedFilePath << std::endl;
    }
    else {
        std::cerr << "File decryption failed!" << std::endl;
        return;
    }

    auto decryptionEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> decryptionDuration = decryptionEnd - decryptionStart;
    std::cout << "Time taken for decryption: " << decryptionDuration.count() << " seconds" << std::endl;
}

int main(int argc, char* argv[]) {
    std::cout << "AES-256 File Encrpytion and Decrpytion\n " << std::endl;
    while (true) {
        int choice;
        std::cout << "Please choose an option: " << std::endl;
        std::cout << "1. Encrypt a file" << std::endl;
        std::cout << "2. Decrypt a file" << std::endl;
        std::cout << "0. Exit" << std::endl;
        std::cin >> choice;

        switch (choice) {
        case 1:
            EncryptProcess();
            break;
        case 2:
            DecryptProcess();
            break;
        case 0:
            std::cout << "Exiting program..." << std::endl;
            return 0;
        default:
            std::cerr << "Invalid choice. Please try again." << std::endl;
            break;
        }
    }

    return 0;
}



